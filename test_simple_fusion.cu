#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <iostream>

int main() {
    std::cout << "🧪 Testing basic CUDA and cuBLAS functionality..." << std::endl;
    
    // Check CUDA device
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices available: " << deviceCount << std::endl;
    
    if (deviceCount == 0) {
        std::cerr << "❌ No CUDA devices found!" << std::endl;
        return 1;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Using device: " << prop.name << std::endl;
    
    // Test basic memory allocation
    const int size = 1024;
    half* d_test;
    hipError_t err = hipMalloc(&d_test, size * sizeof(half));
    if (err != hipSuccess) {
        std::cerr << "❌ CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    // Test cuBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "❌ cuBLAS create failed with status: " << status << std::endl;
        return 1;
    }
    
    std::cout << "✅ Basic CUDA and cuBLAS functionality working!" << std::endl;
    
    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_test);
    
    return 0;
}