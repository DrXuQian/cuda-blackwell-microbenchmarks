#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#ifndef ONEFLOW_CORE_CUDA_SOFTMAX_H_
#define ONEFLOW_CORE_CUDA_SOFTMAX_H_

#include <hipcub/hipcub.hpp>
#include <hip/hip_math_constants.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif  // CUDA_VERSION >= 11000

namespace oneflow {

namespace cuda {

namespace softmax {

constexpr int kWarpSize = 32;

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return max(a, b); }
};

template<template<typename> class ReductionOp, typename T, int thread_group_width = kWarpSize>
__inline__ __device__ T WarpAllReduce(T val) {
  for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
    val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

template<template<typename> class ReductionOp, typename T, int block_size>
__inline__ __device__ T BlockAllReduce(T val) {
  typedef hipcub::BlockReduce<T, block_size> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T result_broadcast;
  T result = BlockReduce(temp_storage).Reduce(val, ReductionOp<T>());
  if (threadIdx.x == 0) { result_broadcast = result; }
  __syncthreads();
  return result_broadcast;
}

template<typename T>
__inline__ __device__ T Inf();

template<>
__inline__ __device__ float Inf<float>() {
  return HIP_INF_F;
}

template<>
__inline__ __device__ double Inf<double>() {
  return HIP_INF;
}

template<typename T>
__inline__ __device__ T Exp(T x);

template<>
__inline__ __device__ float Exp<float>(float x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __expf(x);
#else
  return exp(x);
#endif
}

template<>
__inline__ __device__ double Exp<double>(double x) {
  return exp(x);
}

template<typename T>
__inline__ __device__ T Div(T a, T b);

template<>
__inline__ __device__ float Div<float>(float a, float b) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __fdividef(a, b);
#else
  return a / b;
#endif
}

template<>
__inline__ __device__ double Div<double>(double a, double b) {
  return a / b;
}

template<typename T>
__inline__ __device__ T Log(T x);

template<>
__inline__ __device__ float Log<float>(float x) {
#ifdef OF_SOFTMAX_USE_FAST_MATH
  return __logf(x);
#else
  return log(x);
#endif
}
template<>
__inline__ __device__ double Log<double>(double x) {
  return log(x);
}

inline hipError_t GetNumBlocks(int64_t block_size, int64_t max_blocks, int64_t waves,
                                int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int tpm;
  {
    hipError_t err = hipDeviceGetAttribute(&tpm, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    if (err != hipSuccess) { return err; }
  }
  *num_blocks =
      std::max<int>(1, std::min<int64_t>(max_blocks, sm_count * tpm / block_size * waves));
  return hipSuccess;
}

template<typename T>
struct DefaultComputeType {
  using type = T;
};

template<>
struct DefaultComputeType<half> {
  using type = float;
};

#if CUDA_VERSION >= 11000
template<>
struct DefaultComputeType<hip_bfloat16> {
  using type = float;
};
#endif  // CUDA_VERSION >= 11000

template<typename T, int N>
struct GetPackType {
  using type = typename std::aligned_storage<N * sizeof(T), N * sizeof(T)>::type;
};

template<typename T, int N>
using PackType = typename GetPackType<T, N>::type;

template<typename T, int N>
union Pack {
  static_assert(sizeof(PackType<T, N>) == sizeof(T) * N, "");
  __device__ Pack() {
    // do nothing
  }
  PackType<T, N> storage;
  T elem[N];
};

template<typename SRC, typename DST>
struct DirectLoad {
  DirectLoad(const SRC* src, int64_t row_size) : src(src), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    Pack<SRC, N> pack;
    const int64_t offset = (row * row_size + col) / N;
    pack.storage = *(reinterpret_cast<const PackType<SRC, N>*>(src) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }
  const SRC* src;
  int64_t row_size;
};

template<typename SRC, typename DST>
struct DirectStore {
  DirectStore(DST* dst, int64_t row_size) : dst(dst), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    Pack<DST, N> pack;
    const int64_t offset = (row * row_size + col) / N;
#pragma unroll
    for (int i = 0; i < N; ++i) { pack.elem[i] = static_cast<DST>(src[i]); }
    *(reinterpret_cast<PackType<DST, N>*>(dst) + offset) = pack.storage;
  }
  DST* dst;
  int64_t row_size;
};

enum class Algorithm {
  kSoftmax = 0,
  kLogSoftmax = 1,
};

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, bool padding, Algorithm algorithm>
__global__ void SoftmaxWarpImpl(LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  static_assert(cols_per_thread % pack_size == 0, "");
  static_assert(thread_group_width <= kWarpSize, "");
  static_assert(kWarpSize % thread_group_width == 0, "");
  constexpr int num_packs = cols_per_thread / pack_size;
  assert(cols <= cols_per_thread * thread_group_width);
  ComputeType buf[rows_per_access][cols_per_thread];
  const int global_thread_group_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int num_global_thread_group = gridDim.x * blockDim.y;
  const int lane_id = threadIdx.x;
  const int64_t step = num_global_thread_group * rows_per_access;
  for (int64_t row = global_thread_group_id * rows_per_access; row < rows; row += step) {
    ComputeType thread_max[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_max[row_id] = -Inf<ComputeType>();
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < num_packs; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          load.template load<pack_size>(row_buf + pack_offset, row + row_id, col);
#pragma unroll
          for (int i = 0; i < pack_size; ++i) {
            thread_max[row_id] = max(thread_max[row_id], row_buf[pack_offset + i]);
          }
        } else {
#pragma unroll
          for (int i = 0; i < pack_size; ++i) { row_buf[pack_offset + i] = -Inf<ComputeType>(); }
        }
      }
    }
    ComputeType warp_max[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_max[row_id] = WarpAllReduce<MaxOp, ComputeType, thread_group_width>(thread_max[row_id]);
    }
    ComputeType thread_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_sum[row_id] = 0;
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int i = 0; i < cols_per_thread; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          row_buf[i] = Exp(row_buf[i] - warp_max[row_id]);
          thread_sum[row_id] += row_buf[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          row_buf[i] -= warp_max[row_id];
          thread_sum[row_id] += Exp(row_buf[i]);
        } else {
          __trap();
        }
      }
    }
    ComputeType warp_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_sum[row_id] = WarpAllReduce<SumOp, ComputeType, thread_group_width>(thread_sum[row_id]);
    }
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int i = 0; i < cols_per_thread; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          row_buf[i] = Div(row_buf[i], warp_sum[row_id]);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          row_buf[i] -= Log(warp_sum[row_id]);
        } else {
          __trap();
        }
      }
#pragma unroll
      for (int i = 0; i < num_packs; ++i) {
        const int col = (i * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          store.template store<pack_size>(row_buf + i * pack_size, row + row_id, col);
        }
      }
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, bool padding, Algorithm algorithm>
inline hipError_t LaunchSoftmaxWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                         const int64_t rows, const int64_t cols) {
  constexpr int block_size = 128;
  constexpr int waves = 32;
  static_assert(block_size % thread_group_width == 0, "");
  constexpr int thread_groups_per_block = block_size / thread_group_width;
  dim3 block_dim(thread_group_width, thread_groups_per_block);
  const int64_t num_blocks =
      (rows / rows_per_access + thread_groups_per_block - 1) / thread_groups_per_block;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, num_blocks, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread, thread_group_width,
                  rows_per_access, padding, algorithm>
      <<<grid_dim_x, block_dim, 0, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int cols_per_thread,
         int thread_group_width, int rows_per_access, Algorithm algorithm>
inline hipError_t DispatchSoftmaxWarpImplPadding(hipStream_t stream, LOAD load, STORE store,
                                                  const int64_t rows, const int64_t cols) {
  if (cols == cols_per_thread * thread_group_width) {
    return LaunchSoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread,
                                 thread_group_width, rows_per_access, false, algorithm>(
        stream, load, store, rows, cols);
  } else {
    return LaunchSoftmaxWarpImpl<LOAD, STORE, ComputeType, pack_size, cols_per_thread,
                                 thread_group_width, rows_per_access, true, algorithm>(
        stream, load, store, rows, cols);
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
typename std::enable_if<pack_size == 1, hipError_t>::type DispatchSoftmaxWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                        \
  else if (cols <= (thread_group_width)*pack_size) {                                               \
    if (rows % 2 == 0) {                                                                           \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 2, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    } else {                                                                                       \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 1, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    }                                                                                              \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                      \
  else if (cols <= (col)*kWarpSize) {                                                             \
    return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, col, kWarpSize, 1, \
                                          algorithm>(stream, load, store, rows, cols);            \
  }
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(3)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(5)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(7)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(9)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(11)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(13)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(15)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(17)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(19)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(21)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(23)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(25)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(27)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(29)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(31)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
typename std::enable_if<pack_size == 2, hipError_t>::type DispatchSoftmaxWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                        \
  else if (cols <= (thread_group_width)*pack_size) {                                               \
    if (rows % 2 == 0) {                                                                           \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 2, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    } else {                                                                                       \
      return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size,        \
                                            thread_group_width, 1, algorithm>(stream, load, store, \
                                                                              rows, cols);         \
    }                                                                                              \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                      \
  else if (cols <= (col)*kWarpSize) {                                                             \
    return DispatchSoftmaxWarpImplPadding<LOAD, STORE, ComputeType, pack_size, col, kWarpSize, 1, \
                                          algorithm>(stream, load, store, rows, cols);            \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct DispatchSoftmaxWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols) {
    if (cols % 2 == 0) {
      return DispatchSoftmaxWarpImplCols<LOAD, STORE, ComputeType, 2, algorithm>(stream, load,
                                                                                 store, rows, cols);
    } else {
      return DispatchSoftmaxWarpImplCols<LOAD, STORE, ComputeType, 1, algorithm>(stream, load,
                                                                                 store, rows, cols);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t DispatchSoftmaxWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                           const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxWarpImplPackSize<LOAD, STORE, ComputeType, algorithm>()(stream, load, store,
                                                                                rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
__global__ void SoftmaxBlockSMemImpl(LOAD load, STORE store, const int64_t rows,
                                     const int64_t cols) {
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  auto* buf = reinterpret_cast<ComputeType*>(shared_buf);
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_max = -Inf<ComputeType>();
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        buf[i * num_packs + pack_id] = pack[i];
        thread_max = max(thread_max, pack[i]);
      }
    }
    const ComputeType row_max = BlockAllReduce<MaxOp, ComputeType, block_size>(thread_max);
    ComputeType thread_sum = 0;
    for (int col = tid; col < cols; col += block_size) {
      if (algorithm == Algorithm::kSoftmax) {
        const ComputeType exp_x = Exp(buf[col] - row_max);
        buf[col] = exp_x;
        thread_sum += exp_x;
      } else {
        const ComputeType x = buf[col] - row_max;
        buf[col] = x;
        thread_sum += Exp(x);
      }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = Div(buf[i * num_packs + pack_id], row_sum);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = buf[i * num_packs + pack_id] - Log(row_sum);
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxBlockSMemImpl(hipStream_t stream, LOAD load, STORE store, int smem,
                                              const int64_t rows, const int64_t cols) {
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, smem, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxBlockSMemImplBlockSize(hipStream_t stream, LOAD load,
                                                            STORE store, const int64_t rows,
                                                            const int64_t cols, bool* success) {
  constexpr int block_size_conf_1 = 128;
  constexpr int block_size_conf_2 = 256;
  constexpr int block_size_conf_3 = 512;
  constexpr int block_size_conf_4 = 1024;
  const size_t smem = cols * sizeof(ComputeType);
  int max_active_blocks_conf_1;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_1,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1, algorithm>,
        block_size_conf_1, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_1 <= 0) {
    *success = false;
    return hipSuccess;
  }
  int max_active_blocks_conf_4;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_4,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4, algorithm>,
        block_size_conf_4, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_4 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  int max_active_blocks_conf_3;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_3,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3, algorithm>,
        block_size_conf_3, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_3 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  int max_active_blocks_conf_2;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_2,
        SoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2, algorithm>,
        block_size_conf_2, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_2 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2,
                                      algorithm>(stream, load, store, smem, rows, cols);
  }
  *success = true;
  return LaunchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1,
                                    algorithm>(stream, load, store, smem, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct TryDispatchSoftmaxBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, bool* success) {
    if (cols % 2 == 0) {
      return TryDispatchSoftmaxBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 2, algorithm>(
          stream, load, store, rows, cols, success);
    } else {
      return TryDispatchSoftmaxBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 1, algorithm>(
          stream, load, store, rows, cols, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                   const int64_t rows, const int64_t cols,
                                                   bool* success) {
  return TryDispatchSoftmaxBlockSMemImplPackSize<LOAD, STORE, ComputeType, algorithm>()(
      stream, load, store, rows, cols, success);
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size,
         Algorithm algorithm>
__global__ void SoftmaxBlockUncachedImpl(LOAD load, STORE store, const int64_t rows,
                                         const int64_t cols) {
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_max = -Inf<ComputeType>();
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { thread_max = max(thread_max, pack[i]); }
    }
    const ComputeType row_max = BlockAllReduce<MaxOp, ComputeType, block_size>(thread_max);
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) { thread_sum += Exp(pack[i] - row_max); }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = Div(Exp(pack[i] - row_max), row_sum);
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = (pack[i] - row_max) - Log(row_sum);
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, Algorithm algorithm>
inline hipError_t LaunchSoftmaxBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                  const int64_t rows, const int64_t cols) {
  constexpr int block_size = 1024;
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, 0, stream>>>(load, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
struct DispatchSoftmaxBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols) {
    if (cols % 2 == 0) {
      return LaunchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, 2, algorithm>(
          stream, load, store, rows, cols);
    } else {
      return LaunchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, 1, algorithm>(
          stream, load, store, rows, cols);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType, Algorithm algorithm>
inline hipError_t DispatchSoftmaxBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                    const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImplPackSize<LOAD, STORE, ComputeType, algorithm>()(
      stream, load, store, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                const int64_t cols) {
  if (cols < 1024) {
    return DispatchSoftmaxWarpImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
        stream, load, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
              stream, load, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
          stream, load, store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kSoftmax>(
      stream, load, store, rows, cols);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                   const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxWarpImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
        stream, load, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err =
          TryDispatchSoftmaxBlockSMemImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
              stream, load, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
          stream, load, store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmax(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                   const int64_t cols) {
  return DispatchSoftmaxBlockUncachedImpl<LOAD, STORE, ComputeType, Algorithm::kLogSoftmax>(
      stream, load, store, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, bool padding,
         Algorithm algorithm>
__global__ void SoftmaxGradWarpImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
                                    const int64_t cols) {
  static_assert(cols_per_thread % pack_size == 0, "");
  constexpr int pack_per_thread = cols_per_thread / pack_size;
  assert(cols <= cols_per_thread * thread_group_width);
  static_assert(thread_group_width <= kWarpSize, "");
  static_assert(kWarpSize % thread_group_width == 0, "");
  ComputeType y_buf[rows_per_access][cols_per_thread];
  ComputeType dy_buf[rows_per_access][cols_per_thread];
  const int global_thread_group_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int num_global_thread_group = gridDim.x * blockDim.y;
  const int lane_id = threadIdx.x;
  const int64_t step = num_global_thread_group * rows_per_access;
  for (int64_t row = global_thread_group_id * rows_per_access; row < rows; row += step) {
    ComputeType thread_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_sum[row_id] = 0;
      ComputeType* row_y_buf = y_buf[row_id];
      ComputeType* row_dy_buf = dy_buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < pack_per_thread; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          load_y.template load<pack_size>(row_y_buf + pack_offset, row + row_id, col);
          load_dy.template load<pack_size>(row_dy_buf + pack_offset, row + row_id, col);
#pragma unroll
          for (int i = 0; i < pack_size; ++i) {
            if (algorithm == Algorithm::kSoftmax) {
              thread_sum[row_id] += row_y_buf[pack_offset + i] * row_dy_buf[pack_offset + i];
            } else if (algorithm == Algorithm::kLogSoftmax) {
              thread_sum[row_id] += row_dy_buf[pack_offset + i];
            } else {
              __trap();
            }
          }
        }
      }
    }
    ComputeType warp_sum[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      warp_sum[row_id] = WarpAllReduce<SumOp, ComputeType, thread_group_width>(thread_sum[row_id]);
    }
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      ComputeType* row_y_buf = y_buf[row_id];
      ComputeType* row_dy_buf = dy_buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < pack_per_thread; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          for (int i = 0; i < pack_size; ++i) {
            if (algorithm == Algorithm::kSoftmax) {
              row_dy_buf[pack_offset + i] =
                  (row_dy_buf[pack_offset + i] - warp_sum[row_id]) * row_y_buf[pack_offset + i];
            } else if (algorithm == Algorithm::kLogSoftmax) {
              row_dy_buf[pack_offset + i] -= Exp(row_y_buf[pack_offset + i]) * warp_sum[row_id];
            } else {
              __trap();
            }
          }
          store.template store<pack_size>(row_dy_buf + pack_offset, row + row_id, col);
        }
      }
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, bool padding,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradWarpImpl(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy,
                                             STORE store, const int64_t rows, const int64_t cols) {
  constexpr int block_size = 128;
  constexpr int waves = 32;
  static_assert(block_size % thread_group_width == 0, "");
  constexpr int thread_groups_per_block = block_size / thread_group_width;
  dim3 block_dim(thread_group_width, thread_groups_per_block);
  const int64_t num_blocks =
      (rows / rows_per_access + thread_groups_per_block - 1) / thread_groups_per_block;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, num_blocks, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, cols_per_thread,
                      thread_group_width, rows_per_access, padding, algorithm>
      <<<grid_dim_x, block_dim, 0, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int cols_per_thread, int thread_group_width, int rows_per_access, Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradWarpImplPadding(hipStream_t stream, LOAD_Y load_y,
                                                      LOAD_DY load_dy, STORE store,
                                                      const int64_t rows, const int64_t cols) {
  if (cols == cols_per_thread * thread_group_width) {
    return LaunchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                     cols_per_thread, thread_group_width, rows_per_access, false,
                                     algorithm>(stream, load_y, load_dy, store, rows, cols);
  } else {
    return LaunchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                     cols_per_thread, thread_group_width, rows_per_access, true,
                                     algorithm>(stream, load_y, load_dy, store, rows, cols);
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
typename std::enable_if<pack_size == 1, hipError_t>::type DispatchSoftmaxGradWarpImplCols(
    hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
    const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                     \
  else if (cols <= (thread_group_width)*pack_size) {                                            \
    if (rows % 2 == 0) {                                                                        \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 2, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    } else {                                                                                    \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 1, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    }                                                                                           \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                       \
  else if (cols <= (col)*kWarpSize) {                                                              \
    return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, col, \
                                              kWarpSize, 1, algorithm>(stream, load_y, load_dy,    \
                                                                       store, rows, cols);         \
  }
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(3)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(5)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(7)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(9)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(11)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(13)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(15)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(17)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(19)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(21)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(23)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(25)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(27)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(29)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(31)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
typename std::enable_if<pack_size == 2, hipError_t>::type DispatchSoftmaxGradWarpImplCols(
    hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store, const int64_t rows,
    const int64_t cols) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                     \
  else if (cols <= (thread_group_width)*pack_size) {                                            \
    if (rows % 2 == 0) {                                                                        \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 2, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    } else {                                                                                    \
      return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, \
                                                pack_size, thread_group_width, 1, algorithm>(   \
          stream, load_y, load_dy, store, rows, cols);                                          \
    }                                                                                           \
  }
  DEFINE_ONE_ELIF(1)
  DEFINE_ONE_ELIF(2)
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(col)                                                                       \
  else if (cols <= (col)*kWarpSize) {                                                              \
    return DispatchSoftmaxGradWarpImplPadding<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, col, \
                                              kWarpSize, 1, algorithm>(stream, load_y, load_dy,    \
                                                                       store, rows, cols);         \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(6)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(10)
  DEFINE_ONE_ELIF(12)
  DEFINE_ONE_ELIF(14)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(18)
  DEFINE_ONE_ELIF(20)
  DEFINE_ONE_ELIF(22)
  DEFINE_ONE_ELIF(24)
  DEFINE_ONE_ELIF(26)
  DEFINE_ONE_ELIF(28)
  DEFINE_ONE_ELIF(30)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct DispatchSoftmaxGradWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols) {
    if (cols % 2 == 0) {
      return DispatchSoftmaxGradWarpImplCols<LOAD_Y, LOAD_DY, STORE, ComputeType, 2, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    } else {
      return DispatchSoftmaxGradWarpImplCols<LOAD_Y, LOAD_DY, STORE, ComputeType, 1, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradWarpImpl(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy,
                                               STORE store, const int64_t rows,
                                               const int64_t cols) {
  return DispatchSoftmaxGradWarpImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType, algorithm>()(
      stream, load_y, load_dy, store, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
__global__ void SoftmaxGradBlockSMemImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                                         const int64_t rows, const int64_t cols) {
  extern __shared__ __align__(sizeof(double)) unsigned char grad_shared_buf[];
  auto* y_buf = reinterpret_cast<ComputeType*>(grad_shared_buf);
  auto* dy_buf = y_buf + cols;
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        y_buf[i * num_packs + pack_id] = y_pack[i];
        dy_buf[i * num_packs + pack_id] = dy_pack[i];
        if (algorithm == Algorithm::kSoftmax) {
          thread_sum += y_pack[i] * dy_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          thread_sum += dy_pack[i];
        } else {
          __trap();
        }
      }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          pack[i] = (dy_buf[i * num_packs + pack_id] - row_sum) * y_buf[i * num_packs + pack_id];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          pack[i] = dy_buf[i * num_packs + pack_id] - Exp(y_buf[i * num_packs + pack_id]) * row_sum;
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradBlockSMemImpl(hipStream_t stream, LOAD_Y load_y,
                                                  LOAD_DY load_dy, STORE store, int smem,
                                                  const int64_t rows, const int64_t cols) {
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size, algorithm>
      <<<grid_dim_x, block_size, smem, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxGradBlockSMemImplBlockSize(hipStream_t stream, LOAD_Y load_y,
                                                                LOAD_DY load_dy, STORE store,
                                                                const int64_t rows,
                                                                const int64_t cols, bool* success) {
  constexpr int block_size_conf_1 = 128;
  constexpr int block_size_conf_2 = 256;
  constexpr int block_size_conf_3 = 512;
  constexpr int block_size_conf_4 = 1024;
  const size_t smem = cols * sizeof(ComputeType) * 2;
  int max_active_blocks_conf_1;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_1,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_1,
                                 algorithm>,
        block_size_conf_1, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_1 <= 0) {
    *success = false;
    return hipSuccess;
  }
  int max_active_blocks_conf_4;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_4,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_4,
                                 algorithm>,
        block_size_conf_4, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_4 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_4, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  int max_active_blocks_conf_3;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_3,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_3,
                                 algorithm>,
        block_size_conf_3, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_3 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_3, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  int max_active_blocks_conf_2;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_2,
        SoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size_conf_2,
                                 algorithm>,
        block_size_conf_2, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_2 == max_active_blocks_conf_1) {
    *success = true;
    return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                          block_size_conf_2, algorithm>(stream, load_y, load_dy,
                                                                        store, smem, rows, cols);
  }
  *success = true;
  return LaunchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size,
                                        block_size_conf_1, algorithm>(stream, load_y, load_dy,
                                                                      store, smem, rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct TryDispatchSoftmaxGradBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols, bool* success) {
    if (cols % 2 == 0) {
      return TryDispatchSoftmaxGradBlockSMemImplBlockSize<LOAD_Y, LOAD_DY, STORE, ComputeType, 2,
                                                          algorithm>(stream, load_y, load_dy, store,
                                                                     rows, cols, success);
    } else {
      return TryDispatchSoftmaxGradBlockSMemImplBlockSize<LOAD_Y, LOAD_DY, STORE, ComputeType, 1,
                                                          algorithm>(stream, load_y, load_dy, store,
                                                                     rows, cols, success);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t TryDispatchSoftmaxGradBlockSMemImpl(hipStream_t stream, LOAD_Y load_y,
                                                       LOAD_DY load_dy, STORE store,
                                                       const int64_t rows, const int64_t cols,
                                                       bool* success) {
  return TryDispatchSoftmaxGradBlockSMemImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                     algorithm>()(stream, load_y, load_dy, store,
                                                                  rows, cols, success);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         int block_size, Algorithm algorithm>
__global__ void SoftmaxGradBlockUncachedImpl(LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                                             const int64_t rows, const int64_t cols) {
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = cols / pack_size;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_sum = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);

#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          thread_sum += y_pack[i] * dy_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          thread_sum += dy_pack[i];
        } else {
          __trap();
        }
      }
    }
    const ComputeType row_sum = BlockAllReduce<SumOp, ComputeType, block_size>(thread_sum);
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType y_pack[pack_size];
      ComputeType dy_pack[pack_size];
      load_y.template load<pack_size>(y_pack, row, pack_id * pack_size);
      load_dy.template load<pack_size>(dy_pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        if (algorithm == Algorithm::kSoftmax) {
          dy_pack[i] = (dy_pack[i] - row_sum) * y_pack[i];
        } else if (algorithm == Algorithm::kLogSoftmax) {
          dy_pack[i] -= Exp(y_pack[i]) * row_sum;
        } else {
          __trap();
        }
      }
      store.template store<pack_size>(dy_pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType, int pack_size,
         Algorithm algorithm>
inline hipError_t LaunchSoftmaxGradBlockUncachedImpl(hipStream_t stream, LOAD_Y load_y,
                                                      LOAD_DY load_dy, STORE store,
                                                      const int64_t rows, const int64_t cols) {
  constexpr int block_size = 1024;
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(block_size, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  SoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, pack_size, block_size,
                               algorithm>
      <<<grid_dim_x, block_size, 0, stream>>>(load_y, load_dy, store, rows, cols);
  return hipPeekAtLastError();
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
struct DispatchSoftmaxGradBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                         const int64_t rows, const int64_t cols) {
    if (cols % 2 == 0 && cols > kWarpSize) {
      return LaunchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, 2, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    } else {
      return LaunchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, 1, algorithm>(
          stream, load_y, load_dy, store, rows, cols);
    }
  }
};

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType,
         Algorithm algorithm>
inline hipError_t DispatchSoftmaxGradBlockUncachedImpl(hipStream_t stream, LOAD_Y load_y,
                                                        LOAD_DY load_dy, STORE store,
                                                        const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImplPackSize<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                      algorithm>()(stream, load_y, load_dy, store,
                                                                   rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                    const int64_t rows, const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, Algorithm::kSoftmax>(
        stream, load_y, load_dy, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                            Algorithm::kSoftmax>(
          stream, load_y, load_dy, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                  Algorithm::kSoftmax>(stream, load_y, load_dy,
                                                                       store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                    const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                              Algorithm::kSoftmax>(stream, load_y, load_dy, store,
                                                                   rows, cols);
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                       const int64_t rows, const int64_t cols) {
  if (cols <= 1024) {
    return DispatchSoftmaxGradWarpImpl<LOAD_Y, LOAD_DY, STORE, ComputeType, Algorithm::kLogSoftmax>(
        stream, load_y, load_dy, store, rows, cols);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchSoftmaxGradBlockSMemImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                            Algorithm::kLogSoftmax>(
          stream, load_y, load_dy, store, rows, cols, &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                                  Algorithm::kLogSoftmax>(stream, load_y, load_dy,
                                                                          store, rows, cols);
    }
    return hipSuccess;
  }
}

template<typename LOAD_Y, typename LOAD_DY, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLogSoftmaxGrad(hipStream_t stream, LOAD_Y load_y, LOAD_DY load_dy, STORE store,
                       const int64_t rows, const int64_t cols) {
  return DispatchSoftmaxGradBlockUncachedImpl<LOAD_Y, LOAD_DY, STORE, ComputeType,
                                              Algorithm::kLogSoftmax>(stream, load_y, load_dy,
                                                                      store, rows, cols);
}

}  // namespace softmax

}  // namespace cuda

}  // namespace oneflow

#endif  // ONEFLOW_CORE_CUDA_SOFTMAX_H_

// ============================================================================
// Test harness
// ============================================================================

#include <iostream>
#include <cstring>
#include <cmath>

using namespace oneflow::cuda::softmax;

// Simple load functor for Softmax
template<typename T, typename ComputeType>
struct SimpleLoad {
    using LoadType = T;
    SimpleLoad(const T* ptr, int64_t cols) : ptr(ptr), cols(cols) {}

    template<int N>
    __device__ void load(ComputeType* dst, int64_t row, int64_t col) const {
        for (int i = 0; i < N; i++) {
            dst[i] = static_cast<ComputeType>(ptr[row * cols + col + i]);
        }
    }

    const T* ptr;
    int64_t cols;
};

// Simple store functor for Softmax
template<typename T, typename ComputeType>
struct SimpleStore {
    SimpleStore(T* ptr, int64_t cols) : ptr(ptr), cols(cols) {}

    template<int N>
    __device__ void store(const ComputeType* src, int64_t row, int64_t col) {
        for (int i = 0; i < N; i++) {
            ptr[row * cols + col + i] = static_cast<T>(src[i]);
        }
    }

    T* ptr;
    int64_t cols;
};

void parse_args(int argc, char** argv, int* batch, int* embed, std::string* dtype) {
    *batch = 32;
    *embed = 768;
    *dtype = "float32";

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--batch") == 0 && i + 1 < argc) {
            *batch = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--embed") == 0 && i + 1 < argc) {
            *embed = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--dtype") == 0 && i + 1 < argc) {
            *dtype = argv[++i];
        }
    }
}

template<typename T>
void cpu_softmax(const T* input, T* output, int batch, int embed) {
    for (int b = 0; b < batch; b++) {
        const T* x = input + b * embed;
        T* y = output + b * embed;

        // Find max
        float max_val = static_cast<float>(x[0]);
        for (int i = 1; i < embed; i++) {
            max_val = fmax(max_val, static_cast<float>(x[i]));
        }

        // Compute exp and sum
        float sum = 0.0f;
        for (int i = 0; i < embed; i++) {
            float exp_val = expf(static_cast<float>(x[i]) - max_val);
            y[i] = static_cast<T>(exp_val);
            sum += exp_val;
        }

        // Normalize
        for (int i = 0; i < embed; i++) {
            y[i] = static_cast<T>(static_cast<float>(y[i]) / sum);
        }
    }
}

template<typename T>
void run_test(int batch, int embed) {
    const int size = batch * embed;

    // Allocate host memory
    T* h_input = new T[size];
    T* h_output = new T[size];
    T* h_output_ref = new T[size];

    // Initialize input data
    for (int i = 0; i < size; i++) {
        h_input[i] = static_cast<T>(static_cast<float>(rand()) / RAND_MAX);
    }

    // Allocate device memory
    T *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(T));
    hipMalloc(&d_output, size * sizeof(T));

    // Copy data to device
    hipMemcpy(d_input, h_input, size * sizeof(T), hipMemcpyHostToDevice);

    // Run GPU kernel
    using ComputeType = float;
    SimpleLoad<T, ComputeType> load(d_input, embed);
    SimpleStore<T, ComputeType> store(d_output, embed);

    DispatchSoftmaxWarpImpl<decltype(load), decltype(store), ComputeType, Algorithm::kSoftmax>(
        hipStreamDefault, load, store, batch, embed);

    hipDeviceSynchronize();

    // Copy result back
    hipMemcpy(h_output, d_output, size * sizeof(T), hipMemcpyDeviceToHost);

    // Compute CPU reference
    cpu_softmax(h_input, h_output_ref, batch, embed);

    // Validate
    float max_diff = 0.0f;
    for (int i = 0; i < size; i++) {
        float diff = fabs(static_cast<float>(h_output[i]) - static_cast<float>(h_output_ref[i]));
        max_diff = fmax(max_diff, diff);
    }

    std::cout << "Softmax Test - Batch: " << batch << ", Embed: " << embed << std::endl;
    std::cout << "Max difference: " << max_diff << std::endl;

    if (max_diff < 1e-3f) {
        std::cout << "PASSED" << std::endl;
    } else {
        std::cout << "FAILED" << std::endl;
    }

    // Cleanup
    delete[] h_input;
    delete[] h_output;
    delete[] h_output_ref;
    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char** argv) {
    int batch, embed;
    std::string dtype;

    parse_args(argc, argv, &batch, &embed, &dtype);

    std::cout << "Running Softmax with:" << std::endl;
    std::cout << "  Batch: " << batch << std::endl;
    std::cout << "  Embed: " << embed << std::endl;
    std::cout << "  Dtype: " << dtype << std::endl;
    std::cout << std::endl;

    if (dtype == "float32") {
        run_test<float>(batch, embed);
    } else if (dtype == "float16") {
        run_test<half>(batch, embed);
    } else {
        std::cerr << "Unsupported dtype: " << dtype << std::endl;
        return 1;
    }

    return 0;
}
