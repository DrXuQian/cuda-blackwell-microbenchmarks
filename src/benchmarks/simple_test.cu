#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    std::cout << "Starting simple CUDA test..." << std::endl;
    
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Device 0: " << prop.name << std::endl;
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    }
    
    return 0;
}