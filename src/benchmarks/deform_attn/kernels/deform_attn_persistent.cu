#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <numeric>
#include <chrono>
#include <cfloat>
#include <random>
#include <iostream>
#include <iomanip>
#include <hip/hip_cooperative_groups.h>

#define CUDA_CHECK(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t err, const char *file, int line, bool abort = true) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(err), file, line);
        if (abort) exit(err);
    }
}

#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// Persistent kernel for MS-Deformable Attention
// Each SM gets one thread block that processes multiple queries
// This allows us to use maximum shared memory per block
template <typename scalar_t=__half, const int NUM_POINT=8, const int NUM_LEVELS=4,
          const int CHANNELS=32, const int MAX_SPATIAL_SIZE=20522>
__global__ void ms_deformable_im2col_persistent(
    const scalar_t *data_value,
    const int64_t *data_spatial_shapes,
    const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc,
    const scalar_t *data_attn_weight,
    const int batch_size,
    const int spatial_size,
    const int num_heads,
    const int num_query,
    scalar_t *data_col,
    int *global_counter) {  // Global work counter for work stealing

    // Use maximum shared memory - we can use up to 99KB with opt-in
    extern __shared__ __half shared_mem[];

    // Partition shared memory
    // We'll cache a portion of the value tensor in shared memory
    // With 96KB available, we can store: 96KB / 2 bytes = 48K half values
    // For 32 channels, that's 48K / 32 = 1536 spatial locations
    __half* cached_values = shared_mem;

    const int tid = threadIdx.x;
    const int block_size = blockDim.x;
    const int block_id = blockIdx.x;

    // Persistent kernel loop - keep processing work items
    while (true) {
        // Get next work item using atomic counter
        int work_id = atomicAdd(global_counter, 1);

        // Check if all work is done
        int total_work = batch_size * num_query;
        if (work_id >= total_work) break;

        // Decode work item to batch and query indices
        const int b = work_id / num_query;
        const int q = work_id % num_query;

        // For this query, we'll cache relevant spatial data in shared memory
        // Strategy: Cache the most frequently accessed spatial regions

        // First, load the sampling locations for this query to determine cache strategy
        const int base_loc_idx = ((b * num_query + q) * num_heads) * NUM_LEVELS * NUM_POINT * 2;
        const int base_weight_idx = ((b * num_query + q) * num_heads) * NUM_LEVELS * NUM_POINT;

        // Cooperative loading of value data into shared memory
        // We'll cache data from all levels that this query will access

        // Load spatial shapes first
        __shared__ int64_t s_spatial_shapes[NUM_LEVELS * 2];
        __shared__ int64_t s_level_start_index[NUM_LEVELS];

        if (tid < NUM_LEVELS * 2) {
            s_spatial_shapes[tid] = data_spatial_shapes[tid];
        }
        if (tid < NUM_LEVELS) {
            s_level_start_index[tid] = (tid > 0) ? data_level_start_index[tid - 1] : 0;
        }
        __syncthreads();

        // Analyze sampling locations to determine what to cache
        // For simplicity, we'll cache the first level's data as it's usually the largest
        const int cache_level = 0;  // Cache first level
        const int cache_h = s_spatial_shapes[cache_level * 2];
        const int cache_w = s_spatial_shapes[cache_level * 2 + 1];
        const int cache_spatial_size = cache_h * cache_w;
        const int cache_start_idx = s_level_start_index[cache_level];

        // Calculate how much we can cache
        const int max_cache_elements = 48000;  // ~96KB / 2 bytes per half
        const int cache_elements = min(cache_spatial_size * CHANNELS, max_cache_elements);
        const int cached_spatial_points = cache_elements / CHANNELS;  // How many spatial points we can cache

        // Cooperative loading of cached data
        const int elements_per_thread = (cache_elements + block_size - 1) / block_size;
        const int start_offset = b * spatial_size * CHANNELS + cache_start_idx * CHANNELS;

        #pragma unroll 4
        for (int i = 0; i < elements_per_thread; i++) {
            int idx = tid + i * block_size;
            if (idx < cache_elements) {
                cached_values[idx] = data_value[start_offset + idx];
            }
        }
        __syncthreads();

        // Now each thread processes different channels for this query
        for (int c = tid; c < CHANNELS; c += block_size) {
            scalar_t result = scalar_t(0);

            // Process all levels and points
            #pragma unroll
            for (int l = 0; l < NUM_LEVELS; l++) {
                const int spatial_h = s_spatial_shapes[l * 2];
                const int spatial_w = s_spatial_shapes[l * 2 + 1];
                const int level_start_idx = s_level_start_index[l];

                // Load sampling locations and weights for this level
                __half loc_x[NUM_POINT], loc_y[NUM_POINT], weights[NUM_POINT];

                #pragma unroll
                for (int p = 0; p < NUM_POINT; p++) {
                    const int loc_idx = base_loc_idx + (l * NUM_POINT + p) * 2;
                    const int weight_idx = base_weight_idx + l * NUM_POINT + p;

                    loc_y[p] = data_sampling_loc[loc_idx];
                    loc_x[p] = data_sampling_loc[loc_idx + 1];
                    weights[p] = data_attn_weight[weight_idx];
                }

                // Process each point
                #pragma unroll
                for (int p = 0; p < NUM_POINT; p++) {
                    // Convert normalized coordinates to actual coordinates
                    const float y = (__half2float(loc_y[p]) + 1) * spatial_h / 2.0f - 0.5f;
                    const float x = (__half2float(loc_x[p]) + 1) * spatial_w / 2.0f - 0.5f;

                    // Check bounds
                    if (y > -1 && x > -1 && y < spatial_h && x < spatial_w) {
                        const int y_low = floorf(y);
                        const int x_low = floorf(x);
                        const int y_high = y_low + 1;
                        const int x_high = x_low + 1;

                        const __half ly = __float2half(y - y_low);
                        const __half lx = __float2half(x - x_low);
                        const __half hy = __float2half(1 - (y - y_low));
                        const __half hx = __float2half(1 - (x - x_low));

                        // Bilinear interpolation
                        scalar_t val = scalar_t(0);

                        // Helper function to read value (from cache if possible, otherwise from global)
                        auto read_value = [&](int y_coord, int x_coord) -> scalar_t {
                            if (y_coord >= 0 && x_coord >= 0 && y_coord < spatial_h && x_coord < spatial_w) {
                                int spatial_idx = y_coord * spatial_w + x_coord;

                                // Check if this is in our cached level and within cached range
                                if (l == cache_level && spatial_idx < cached_spatial_points) {
                                    // Read from shared memory
                                    int cache_idx = spatial_idx * CHANNELS + c;
                                    return cached_values[cache_idx];
                                }

                                // Fall back to global memory
                                const int global_idx = (b * spatial_size + level_start_idx + spatial_idx) * CHANNELS + c;
                                return data_value[global_idx];
                            }
                            return scalar_t(0);
                        };

                        // Perform bilinear interpolation using cached or global memory
                        val = __hfma(read_value(y_low, x_low), __hmul(hy, hx), val);
                        val = __hfma(read_value(y_low, x_high), __hmul(hy, lx), val);
                        val = __hfma(read_value(y_high, x_low), __hmul(ly, hx), val);
                        val = __hfma(read_value(y_high, x_high), __hmul(ly, lx), val);

                        result = __hfma(weights[p], val, result);
                    }
                }
            }

            // Write output
            const int out_idx = (b * num_query + q) * num_heads * CHANNELS + c;
            data_col[out_idx] = result;
        }

        __syncthreads();  // Ensure all threads finish before getting next work item
    }
}

int main() {
    std::cout << "=== Persistent Kernel MS-Deformable Attention ===" << std::endl;

    // Check device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device: " << prop.name << std::endl;
    std::cout << "SMs: " << prop.multiProcessorCount << std::endl;
    std::cout << "Max shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "Max shared memory per SM: " << prop.sharedMemPerMultiprocessor << " bytes" << std::endl;

    // Configuration - now we can use larger spatial sizes!
    const int batch = 8;
    const int num_heads = 1;
    const int channels = 32;
    const int num_levels = 4;
    const int num_points = 8;
    const int num_query = 1024;  // More queries

    // Larger feature maps since we have more shared memory
    const std::vector<int64_t> h_spatial_shapes = {
        64, 64,   // Level 0: 64x64 = 4096
        32, 32,   // Level 1: 32x32 = 1024
        16, 16,   // Level 2: 16x16 = 256
        8, 8      // Level 3: 8x8 = 64
    };

    // Calculate spatial size and level start indices
    int spatial_size = 0;
    std::vector<int64_t> h_level_start_index;

    for (int i = 0; i < num_levels; i++) {
        if (i > 0) h_level_start_index.push_back(spatial_size);
        int h = h_spatial_shapes[i * 2];
        int w = h_spatial_shapes[i * 2 + 1];
        spatial_size += h * w;
    }

    std::cout << "\nConfiguration:" << std::endl;
    std::cout << "  Batch size: " << batch << std::endl;
    std::cout << "  Spatial size: " << spatial_size << std::endl;
    std::cout << "  Num queries: " << num_query << std::endl;
    std::cout << "  Channels: " << channels << std::endl;
    std::cout << "  Level shapes: ";
    for (int i = 0; i < num_levels; i++) {
        std::cout << h_spatial_shapes[i*2] << "x" << h_spatial_shapes[i*2+1];
        if (i < num_levels - 1) std::cout << ", ";
    }
    std::cout << std::endl;

    // Allocate host memory
    const int64_t value_size = batch * spatial_size * channels;
    const int64_t output_size = batch * num_query * num_heads * channels;
    const int64_t sampling_loc_size = batch * num_query * num_heads * num_levels * num_points * 2;
    const int64_t attn_weight_size = batch * num_query * num_heads * num_levels * num_points;

    std::vector<__half> h_value(value_size);
    std::vector<__half> h_sampling_loc(sampling_loc_size);
    std::vector<__half> h_attn_weight(attn_weight_size);
    std::vector<__half> h_output(output_size);

    // Initialize data
    std::cout << "Initializing data..." << std::endl;

    // Initialize value tensor
    for (size_t i = 0; i < h_value.size(); i++) {
        h_value[i] = __float2half(sinf(i * 0.001f) * 0.5f + 0.5f);
    }

    // Initialize sampling locations
    std::mt19937 gen(42);
    std::uniform_real_distribution<float> loc_dist(-0.8f, 0.8f);
    for (size_t i = 0; i < h_sampling_loc.size(); i++) {
        h_sampling_loc[i] = __float2half(loc_dist(gen));
    }

    // Initialize attention weights (normalized)
    for (int b = 0; b < batch; b++) {
        for (int q = 0; q < num_query; q++) {
            for (int h = 0; h < num_heads; h++) {
                float sum = 0.0f;
                int base = ((b * num_query + q) * num_heads + h) * num_levels * num_points;

                for (int i = 0; i < num_levels * num_points; i++) {
                    float w = expf(-i * 0.1f);
                    h_attn_weight[base + i] = __float2half(w);
                    sum += w;
                }

                for (int i = 0; i < num_levels * num_points; i++) {
                    h_attn_weight[base + i] = __float2half(__half2float(h_attn_weight[base + i]) / sum);
                }
            }
        }
    }

    // Allocate device memory
    __half *d_value, *d_sampling_loc, *d_attn_weight, *d_output;
    int64_t *d_spatial_shapes, *d_level_start_index;
    int *d_global_counter;

    CUDA_CHECK(hipMalloc(&d_value, value_size * sizeof(__half)));
    CUDA_CHECK(hipMalloc(&d_sampling_loc, sampling_loc_size * sizeof(__half)));
    CUDA_CHECK(hipMalloc(&d_attn_weight, attn_weight_size * sizeof(__half)));
    CUDA_CHECK(hipMalloc(&d_output, output_size * sizeof(__half)));
    CUDA_CHECK(hipMalloc(&d_spatial_shapes, h_spatial_shapes.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_level_start_index, h_level_start_index.size() * sizeof(int64_t)));
    CUDA_CHECK(hipMalloc(&d_global_counter, sizeof(int)));

    // Copy to device
    CUDA_CHECK(hipMemcpy(d_value, h_value.data(), value_size * sizeof(__half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sampling_loc, h_sampling_loc.data(), sampling_loc_size * sizeof(__half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_attn_weight, h_attn_weight.data(), attn_weight_size * sizeof(__half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_spatial_shapes, h_spatial_shapes.data(), h_spatial_shapes.size() * sizeof(int64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_level_start_index, h_level_start_index.data(), h_level_start_index.size() * sizeof(int64_t), hipMemcpyHostToDevice));

    // Launch configuration for persistent kernel
    // One thread block per SM for maximum shared memory usage
    const int num_blocks = prop.multiProcessorCount;
    const int threads_per_block = 256;  // Good occupancy

    // Calculate shared memory size - use maximum available
    size_t smem_size = 96 * 1024;  // 96KB - close to the 99KB limit

    // Set maximum dynamic shared memory
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        ms_deformable_im2col_persistent<__half), 8, 4, 32, 20522>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size));

    std::cout << "\nLaunching persistent kernel:" << std::endl;
    std::cout << "  Blocks: " << num_blocks << " (one per SM)" << std::endl;
    std::cout << "  Threads per block: " << threads_per_block << std::endl;
    std::cout << "  Shared memory per block: " << smem_size / 1024.0 << " KB" << std::endl;
    std::cout << "  Total work items: " << batch * num_query << std::endl;

    // Warmup
    std::cout << "Running warmup..." << std::endl;
    for (int i = 0; i < 5; i++) {
        CUDA_CHECK(hipMemset(d_global_counter, 0, sizeof(int)));
        ms_deformable_im2col_persistent<__half, 8, 4, 32, 20522>
            <<<num_blocks, threads_per_block, smem_size>>>(
            d_value, d_spatial_shapes, d_level_start_index,
            d_sampling_loc, d_attn_weight,
            batch, spatial_size, num_heads, num_query,
            d_output, d_global_counter);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Benchmark
    std::cout << "Running benchmark..." << std::endl;
    const int num_iterations = 100;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    for (int i = 0; i < num_iterations; i++) {
        CUDA_CHECK(hipMemset(d_global_counter, 0, sizeof(int)));
        ms_deformable_im2col_persistent<__half, 8, 4, 32, 20522>
            <<<num_blocks, threads_per_block, smem_size>>>(
            d_value, d_spatial_shapes, d_level_start_index,
            d_sampling_loc, d_attn_weight,
            batch, spatial_size, num_heads, num_query,
            d_output, d_global_counter);
    }
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout << "\n=== Performance Results ===" << std::endl;
    std::cout << "Total time: " << milliseconds << " ms" << std::endl;
    std::cout << "Average kernel time: " << milliseconds / num_iterations * 1000 << " microseconds" << std::endl;
    std::cout << "Throughput: " << (num_iterations * 1000.0f) / milliseconds << " iterations/second" << std::endl;

    // Calculate GFLOPS
    double ops_per_output = num_levels * num_points * 10;
    double total_ops = batch * num_query * num_heads * channels * ops_per_output;
    double gflops = (total_ops * num_iterations) / (milliseconds * 1e6);
    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;

    // Copy back and verify
    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, output_size * sizeof(__half), hipMemcpyDeviceToHost));

    // Check results
    std::cout << "\nFirst 20 output values:" << std::endl;
    for (int i = 0; i < std::min(20LL, (long long)output_size); i++) {
        if (i % 10 == 0 && i > 0) std::cout << std::endl;
        std::cout << std::fixed << std::setprecision(4) << __half2float(h_output[i]) << " ";
    }
    std::cout << std::endl;

    // Validation
    bool has_nonzero = false;
    float max_val = 0.0f;
    float min_val = FLT_MAX;
    float sum_val = 0.0f;

    for (int i = 0; i < output_size; i++) {
        float val = __half2float(h_output[i]);
        if (val != 0.0f) has_nonzero = true;
        max_val = fmaxf(max_val, val);
        min_val = fminf(min_val, val);
        sum_val += val;
    }

    std::cout << "\n=== Validation ===" << std::endl;
    std::cout << "Has non-zero values: " << (has_nonzero ? "Yes" : "No") << std::endl;
    std::cout << "Min value: " << min_val << std::endl;
    std::cout << "Max value: " << max_val << std::endl;
    std::cout << "Mean value: " << sum_val / output_size << std::endl;

    // Cleanup
    hipFree(d_value);
    hipFree(d_sampling_loc);
    hipFree(d_attn_weight);
    hipFree(d_output);
    hipFree(d_spatial_shapes);
    hipFree(d_level_start_index);
    hipFree(d_global_counter);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "\n✅ Persistent kernel test completed successfully!" << std::endl;

    return 0;
}