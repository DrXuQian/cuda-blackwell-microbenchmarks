#include "hip/hip_runtime.h"
#include "common.h"

__device__ void copy_gmem_to_shmem_warp(half* shmem_ptr, const half* gmem_ptr, 
                                        int rows, int cols, int src_ld, int warp_id) {
    const int lane_id = threadIdx.x % WARP_SIZE;
    
    // Only specified warp performs the copy
    if (threadIdx.x >= warp_id * WARP_SIZE && threadIdx.x < (warp_id + 1) * WARP_SIZE) {
        const int total_elements = rows * cols;
        for (int i = lane_id; i < total_elements; i += WARP_SIZE) {
            int row = i / cols;
            int col = i % cols;
            shmem_ptr[i] = gmem_ptr[row * src_ld + col];
        }
    }
}

__device__ void copy_gmem_to_shmem(half* shmem_ptr, const half* gmem_ptr, 
                                   int rows, int cols, int src_ld) {
    const int tid = threadIdx.x;
    const int block_size = blockDim.x;
    
    // All threads participate in copy
    const int total_elements = rows * cols;
    for (int i = tid; i < total_elements; i += block_size) {
        int row = i / cols;
        int col = i % cols;
        shmem_ptr[i] = gmem_ptr[row * src_ld + col];
    }
}

__global__ void warp_specialized_mma_kernel(
    const half* A, const half* B, float* C,
    int M_dim, int N_dim, int K_dim
) {
    extern __shared__ char shmem[];
    
    const int warp_id = threadIdx.x / WARP_SIZE;
    
    half* shmem_A = reinterpret_cast<half*>(shmem);
    half* shmem_B = shmem_A + 2 * M * K;  // Account for double buffering
    
    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> frag_A;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::row_major> frag_B;
    wmma::fragment<wmma::accumulator, M, N, K, float> frag_C;
    
    wmma::fill_fragment(frag_C, 0.0f);
    
    int block_row = blockIdx.y * M;
    int block_col = blockIdx.x * N;
    
    // Create double buffer for true async operation
    half* shmem_A_buffers[2] = {shmem_A, shmem_A + M * K};
    half* shmem_B_buffers[2] = {shmem_B, shmem_B + K * N};
    int buffer_idx = 0;
    
    // Pre-load first chunk
    copy_gmem_to_shmem(shmem_A_buffers[0], &A[block_row * K_dim], M, K, K_dim);
    copy_gmem_to_shmem(shmem_B_buffers[0], &B[block_col], K, N, N_dim);
    __syncthreads();
    
    for (int k = 0; k < K_dim; k += K) {
        int current_buffer = buffer_idx;
        int next_buffer = 1 - buffer_idx;
        
        // Warp specialization: Warp 0 loads next data while Warp 1+ computes
        if (warp_id == 0 && k + K < K_dim) {
            // Warp 0: Load next chunk asynchronously
            copy_gmem_to_shmem_warp(shmem_A_buffers[next_buffer], 
                                   &A[block_row * K_dim + k + K], M, K, K_dim, 0);
            copy_gmem_to_shmem_warp(shmem_B_buffers[next_buffer], 
                                   &B[(k + K) * N_dim + block_col], K, N, N_dim, 0);
        } else if (warp_id == 1) {
            // Warp 1: Perform computation on current data
            wmma::load_matrix_sync(frag_A, shmem_A_buffers[current_buffer], K);
            wmma::load_matrix_sync(frag_B, shmem_B_buffers[current_buffer], N);
            wmma::mma_sync(frag_C, frag_A, frag_B, frag_C);
        }
        
        __syncthreads();
        buffer_idx = next_buffer;
    }
    
    if (warp_id == 1) {
        wmma::store_matrix_sync(&C[block_row * N_dim + block_col], frag_C, N_dim, wmma::mem_row_major);
    }
}

int main() {
    printf("Warp Specialized MMA Kernel Performance Test\n");
    printf("===========================================\n");
    
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device: %s (Compute Capability %d.%d)\n\n", prop.name, prop.major, prop.minor);
    
    const int M_dim = 1024, N_dim = 1024, K_dim = 1024;
    
    size_t size_A = M_dim * K_dim * sizeof(half);
    size_t size_B = K_dim * N_dim * sizeof(half);
    size_t size_C = M_dim * N_dim * sizeof(float);
    
    half *h_A = (half*)malloc(size_A);
    half *h_B = (half*)malloc(size_B);
    float *h_C_gpu = (float*)malloc(size_C);
    
    if (!h_A || !h_B || !h_C_gpu) {
        printf("Failed to allocate host memory\n");
        return 1;
    }
    
    // Initialize with small random values
    srand(42);
    for (int i = 0; i < M_dim * K_dim; i++) {
        h_A[i] = __float2half((static_cast<float>(rand()) / RAND_MAX - 0.5f) * 0.1f);
    }
    for (int i = 0; i < K_dim * N_dim; i++) {
        h_B[i] = __float2half((static_cast<float>(rand()) / RAND_MAX - 0.5f) * 0.1f);
    }
    
    half *d_A, *d_B;
    float *d_C;
    
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    printf("Testing Warp Specialized MMA Kernel:\n");
    size_t shmem_size = 4 * (M * K + K * N) * sizeof(half); // Double buffer
    hipMemset(d_C, 0, size_C);
    benchmark_kernel(warp_specialized_mma_kernel, "Warp Specialized MMA", d_A, d_B, d_C, M_dim, N_dim, K_dim, shmem_size);
    
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);
    
    AccuracyResult acc_result;
    if (verify_with_cublas(h_A, h_B, h_C_gpu, M_dim, N_dim, K_dim, &acc_result)) {
        printf("\n");
        print_accuracy_result(acc_result, "Warp Specialized MMA");
    } else {
        printf("\n");
        print_accuracy_result(acc_result, "Warp Specialized MMA");
    }
    
    printf("\ncuBLAS Reference Performance:\n");
    benchmark_cublas(d_A, d_B, d_C, M_dim, N_dim, K_dim);
    
    free(h_A);
    free(h_B);
    free(h_C_gpu);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    printf("\n🎉 Warp specialized kernel test completed!\n");
    return 0;
}