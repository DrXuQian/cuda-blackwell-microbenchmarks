#include "hip/hip_runtime.h"
#include "common.h"

__device__ void copy_gmem_to_shmem_optimized(half* shmem_ptr, const half* gmem_ptr, 
                                            int rows, int cols, int src_ld) {
    const int tid = threadIdx.x;
    const int block_size = blockDim.x;
    
    // Simple coalesced copy - handle row-major to contiguous conversion
    const int total_elements = rows * cols;
    for (int i = tid; i < total_elements; i += block_size) {
        int row = i / cols;
        int col = i % cols;
        shmem_ptr[i] = gmem_ptr[row * src_ld + col];
    }
}

// Multi-warp specialized kernel - better work distribution
__global__ void warp_specialized_mma_kernel_optimized(
    const half* A, const half* B, float* C,
    int M_dim, int N_dim, int K_dim
) {
    extern __shared__ char shmem[];
    
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    
    half* shmem_A = reinterpret_cast<half*>(shmem);
    half* shmem_B = shmem_A + 2 * M * K;  // Double buffer A
    
    // Create double buffers for overlapped execution
    half* shmem_A_buffers[2] = {shmem_A, shmem_A + M * K};
    half* shmem_B_buffers[2] = {shmem_B, shmem_B + K * N};
    
    // Multiple compute fragments for better parallelism
    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> frag_A[2];
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::row_major> frag_B[2];
    wmma::fragment<wmma::accumulator, M, N, K, float> frag_C[2];
    
    // Initialize accumulators
    for (int i = 0; i < 2; i++) {
        wmma::fill_fragment(frag_C[i], 0.0f);
    }
    
    int block_row = blockIdx.y * M;
    int block_col = blockIdx.x * N;
    
    int buffer_idx = 0;
    
    // Pre-load first chunk with all warps participating
    copy_gmem_to_shmem_optimized(shmem_A_buffers[0], &A[block_row * K_dim], M, K, K_dim);
    copy_gmem_to_shmem_optimized(shmem_B_buffers[0], &B[block_col], K, N, N_dim);
    __syncthreads();
    
    for (int k = 0; k < K_dim; k += K) {
        int current_buffer = buffer_idx;
        int next_buffer = 1 - buffer_idx;
        
        // Improved work distribution:
        // Warps 0-1: Handle data loading (50% of warps)
        // Warps 2-3: Handle computation (50% of warps)  
        if (warp_id < 2 && k + K < K_dim) {
            // Data loading warps - split work between two warps
            if (warp_id == 0) {
                // Warp 0: Load A matrix
                copy_gmem_to_shmem_optimized(shmem_A_buffers[next_buffer], 
                                           &A[block_row * K_dim + k + K], M, K, K_dim);
            } else {
                // Warp 1: Load B matrix  
                copy_gmem_to_shmem_optimized(shmem_B_buffers[next_buffer], 
                                           &B[(k + K) * N_dim + block_col], K, N, N_dim);
            }
        } else if (warp_id == 2) {
            // Single compute warp performs MMA operations
            wmma::load_matrix_sync(frag_A[0], shmem_A_buffers[current_buffer], K);
            wmma::load_matrix_sync(frag_B[0], shmem_B_buffers[current_buffer], N);
            wmma::mma_sync(frag_C[0], frag_A[0], frag_B[0], frag_C[0]);
        }
        
        __syncthreads();
        buffer_idx = next_buffer;
    }
    
    // Only use one compute warp to avoid double computation
    if (warp_id == 2) {
        // Primary compute warp stores results
        wmma::store_matrix_sync(&C[block_row * N_dim + block_col], frag_C[0], N_dim, wmma::mem_row_major);
    }
}

int main() {
    printf("Optimized Warp Specialized MMA Kernel Performance Test\n");
    printf("======================================================\n");
    
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device: %s (Compute Capability %d.%d)\n\n", prop.name, prop.major, prop.minor);
    
    const int M_dim = 1024, N_dim = 1024, K_dim = 1024;
    
    size_t size_A = M_dim * K_dim * sizeof(half);
    size_t size_B = K_dim * N_dim * sizeof(half);
    size_t size_C = M_dim * N_dim * sizeof(float);
    
    half *h_A = (half*)malloc(size_A);
    half *h_B = (half*)malloc(size_B);
    float *h_C_gpu = (float*)malloc(size_C);
    
    if (!h_A || !h_B || !h_C_gpu) {
        printf("Failed to allocate host memory\n");
        return 1;
    }
    
    // Initialize with small random values
    srand(42);
    for (int i = 0; i < M_dim * K_dim; i++) {
        h_A[i] = __float2half((static_cast<float>(rand()) / RAND_MAX - 0.5f) * 0.1f);
    }
    for (int i = 0; i < K_dim * N_dim; i++) {
        h_B[i] = __float2half((static_cast<float>(rand()) / RAND_MAX - 0.5f) * 0.1f);
    }
    
    half *d_A, *d_B;
    float *d_C;
    
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);
    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    printf("Testing Optimized Warp Specialized MMA Kernel:\n");
    size_t shmem_size = 4 * (M * K + K * N) * sizeof(half); // Double buffer
    hipMemset(d_C, 0, size_C);
    benchmark_kernel(warp_specialized_mma_kernel_optimized, "Optimized Warp Specialized MMA", 
                     d_A, d_B, d_C, M_dim, N_dim, K_dim, shmem_size);
    
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);
    
    AccuracyResult acc_result;
    if (verify_with_cublas(h_A, h_B, h_C_gpu, M_dim, N_dim, K_dim, &acc_result)) {
        printf("\n");
        print_accuracy_result(acc_result, "Optimized Warp Specialized MMA");
    } else {
        printf("\n");
        print_accuracy_result(acc_result, "Optimized Warp Specialized MMA");
    }
    
    printf("\ncuBLAS Reference Performance:\n");
    benchmark_cublas(d_A, d_B, d_C, M_dim, N_dim, K_dim);
    
    free(h_A);
    free(h_B);
    free(h_C_gpu);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    printf("\n🎉 Optimized warp specialized kernel test completed!\n");
    return 0;
}